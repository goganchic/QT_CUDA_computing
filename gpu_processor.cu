
#include <hip/hip_runtime.h>
#define DATA_SIZE 10000000
#define BLOCKS_COUNT 1
#define THREADS_COUNT 64

__global__ void findMean(unsigned int dataForBlock, float *inputData, float *results)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float result = 0;

    for (int i = 0; i < dataForBlock; i++)
    {
        result += inputData[index * dataForBlock + i];
    }

    result /= dataForBlock;
    results[index] = result;
}

void processWithGPU(float *blocks, float *results, unsigned int blockSize, unsigned int blocksCount)
{
    unsigned int realDataCount = blockSize * blocksCount;
    hipSetDevice(0);
    float *deviceInputData, *deviceResults;

    hipMalloc((void **)&deviceInputData, realDataCount * sizeof(float));
    hipMalloc((void **)&deviceResults, realDataCount * sizeof(float));
    hipMemcpy(deviceInputData, blocks, realDataCount * sizeof(float), hipMemcpyHostToDevice);

    findMean<<<1, blocksCount>>>(blockSize, deviceInputData, deviceResults);

    hipMemcpy((void *)results, deviceResults, blocksCount * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInputData);
    hipFree(deviceResults);
}
